#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2019 by Wuwei Lin
 * \file proposal.cu
 * \brief Proposal
 */
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file proposal.cu
 * \brief Proposal Operator
 * \author Shaoqing Ren, Jian Guo, Pengfei Chen, Yuntao Chen
*/
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>

#include <tvm/relay/base.h>
#include <math.h>
#include <map>
#include <vector>
#include <string>
#include <utility>
#include <ctime>
#include <iostream>

#include "./decode_BBox.h"
#include "./decode_BBox-inl.h"

#define THREAD_PER_BLOCK 256
#define FRCNN_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
} while (0)

using namespace mshadow::cuda;

namespace tvm {
namespace contrib {
namespace mxnet {


using tvm::relay::IndexExpr;

// bbox prediction and clip to the image borders
template<typename DType>
__global__ void BBoxTransformInv(DType* boxes,
                                 DType* bbox_deltas,
                                 const int count,
                                 const int num_class,
                                 const int boxes_1,const int boxes_2,
                                 const int bbox_deltas_1,const int bbox_deltas_2,
                                 const int im_info_1,
                                 const int out_1,const int out_2,
                                 DType* bbox_mean,
                                 DType* bbox_std,
                                 const bool class_agnostic,
                                 const DType* im_info,
                                 DType* out) {

  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  int cidx = bidx*THREAD_PER_BLOCK+tidx;
  
  //compute if cidx is less than count
  if(cidx<count){
      int n = cidx/(boxes_1*num_class);
      int index = cidx%(boxes_1*num_class)/num_class;
      int cls = cidx%num_class;
      int offset = n*boxes_1*boxes_2+index*boxes_2;
      float width = boxes[offset+2] - boxes[offset] + 1.0f;
      float height = boxes[offset+3] - boxes[offset+1] + 1.0f;
      float ctr_x = boxes[offset] + 0.5f * (width - 1.0f);
      float ctr_y = boxes[offset+1] + 0.5f * (height - 1.0f);

      int decode_cls = class_agnostic ? 1 : cls;
      offset = n*bbox_deltas_1*bbox_deltas_2+index*bbox_deltas_2;
      float dx = bbox_deltas[offset+decode_cls*4+0] * bbox_std[0] + bbox_mean[0];
      float dy = bbox_deltas[offset+decode_cls*4+1] * bbox_std[1] + bbox_mean[1];
      float dw = bbox_deltas[offset+decode_cls*4+2] * bbox_std[2] + bbox_mean[2];
      float dh = bbox_deltas[offset+decode_cls*4+3] * bbox_std[3] + bbox_mean[3];

      float pred_ctr_x = dx * width + ctr_x;
      float pred_ctr_y = dy * height + ctr_y;
      float pred_w = ::exp(dw) * width;
      float pred_h = ::exp(dh) * height;

      float pred_x1 = pred_ctr_x - 0.5f * (pred_w - 1.0f);
      float pred_y1 = pred_ctr_y - 0.5f * (pred_h - 1.0f);
      float pred_x2 = pred_ctr_x + 0.5f * (pred_w - 1.0f);
      float pred_y2 = pred_ctr_y + 0.5f * (pred_h - 1.0f);
      
      offset = n*im_info_1;
      pred_x1 = pred_x1<im_info[offset+1] - 1.0f?pred_x1:im_info[offset+1] - 1.0f;
      pred_y1 = pred_y1<im_info[offset+0] - 1.0f?pred_y1:im_info[offset+0] - 1.0f;
      pred_x2 = pred_x2<im_info[offset+1] - 1.0f?pred_x2:im_info[offset+1] - 1.0f;
      pred_y2 = pred_y2<im_info[offset+0] - 1.0f?pred_y2:im_info[offset+0] - 1.0f;

      pred_x1 = pred_x1>0.0f?pred_x1:0.0f;
      pred_y1 = pred_y1>0.0f?pred_y1:0.0f;
      pred_x2 = pred_x2>0.0f?pred_x2:0.0f;
      pred_y2 = pred_y2>0.0f?pred_y2:0.0f;
      
      offset = n*out_1*out_2+index*out_2;
      out[offset+cls*4+0] = pred_x1;
      out[offset+cls*4+1] = pred_y1;
      out[offset+cls*4+2] = pred_x2;
      out[offset+cls*4+3] = pred_y2;
  }

}

Decode_BBoxOp::Decode_BBoxOp(const Decode_BBoxSign& param) {
    auto boxes_shape = param.rois->shape;
    nbatch = boxes_shape[0];
    class_agnostic = param.class_agnostic;
    bbox_mean = std::move(param.bbox_mean);
    bbox_std = std::move(param.bbox_std);
    bbox_mean_gpu = mshadow::NewTensor<gpu, float, 1>(Shape1(4), 0.);
    bbox_std_gpu = mshadow::NewTensor<gpu, float, 1>(Shape1(4), 0);
}

void Decode_BBoxOp::Forward(
             mshadow::Tensor<gpu, 3, float>& boxes,
             mshadow::Tensor<gpu, 3, float>& bbox_deltas,
             mshadow::Tensor<gpu, 2, float>& im_info,
             mshadow::Tensor<gpu, 3, float>& out) {
  //copy bbox_mean and bbox_std to gpu
  FRCNN_CUDA_CHECK(hipMemcpy(bbox_mean.data(),
                              bbox_mean_gpu.dptr_,
                              sizeof(float) * bbox_mean.size(),
                              hipMemcpyHostToDevice));
    
  
  FRCNN_CUDA_CHECK(hipMemcpy(bbox_std.data(),
                              bbox_std_gpu.dptr_,
                              sizeof(float) * bbox_std.size(),
                              hipMemcpyHostToDevice));

  //decode bbox
  int boxes_1 = boxes.size(1);
  int boxes_2 = boxes.size(2);
  int bbox_deltas_1 = bbox_deltas.size(0);
  int bbox_deltas_2 = bbox_deltas.size(0);
  int im_info_1= im_info.size(1);
  int num_class = class_agnostic ? 1 : (bbox_deltas.size(2) / 4);
  int count = nbatch*boxes_1*num_class;

  dim3 dimGrid((count + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
  dim3 dimBlock(THREAD_PER_BLOCK);
  BBoxTransformInv<<<dimGrid, dimBlock>>>(boxes.dptr_, bbox_deltas.dptr_, count,num_class,
                                          boxes_1,boxes_2,bbox_deltas_1,bbox_deltas_2,im_info_1,
                                          bbox_mean_gpu.dptr_, bbox_std_gpu.dptr_, class_agnostic,
                                          im_info.dptr_, out.dptr_);

  FRCNN_CUDA_CHECK(hipPeekAtLastError());
  }
}

ProposalGPUOp::~ProposalGPUOp() {
  mshadow::FreeSpace(&bbox_mean_gpu);
  mshadow::FreeSpace(&bbox_std_gpu);
}


}  // namespace mxnet
}  // namespace contrib
}  // namespace tvm
